
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// CUDA code to multiply matrices
__global__
void multiply(int* A, int* B, int* C, int size) {
    // Uses thread indices and block indices to compute each element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        int sum = 0;
        for (int i = 0; i < size; i++) {
            sum += A[row * size + i] * B[i * size + col];
        }
        C[row * size + col] = sum;
    }
}

void initialize(int* matrix, int size) {
    for (int i = 0; i < size * size; i++) {
        matrix[i] = rand() % 10;
    }
}

void print(int* matrix, int size) {
    for (int row = 0; row < size; row++) {
        for (int col = 0; col < size; col++) {
            printf("%d ", matrix[row * size + col]);
        }
        printf("\n");
    }
    printf("\n");
}

int main() {
    int* A, * B, * C;

    int N = 2;
    int blockSize = 16;

    int matrixSize = N * N;
    size_t matrixBytes = matrixSize * sizeof(int);

    A = (int*)malloc(matrixBytes);
    B = (int*)malloc(matrixBytes);
    C = (int*)malloc(matrixBytes);

    initialize(A, N);
    initialize(B, N);
    printf("Matrix A: \n");
    print(A, N);

    printf("Matrix B: \n");
    print(B, N);

    int* X, * Y, * Z;
    // Allocate space
    hipMalloc(&X, matrixBytes);
    hipMalloc(&Y, matrixBytes);
    hipMalloc(&Z, matrixBytes);

    // Copy values from A to X
    hipMemcpy(X, A, matrixBytes, hipMemcpyHostToDevice);

    // Copy values from A to X and B to Y
    hipMemcpy(Y, B, matrixBytes, hipMemcpyHostToDevice);

    // Threads per CTA dimension
    int THREADS = 2;

    // Blocks per grid dimension (assumes THREADS divides N evenly)
    int BLOCKS = N / THREADS;

    // Launch kernel
    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);
    multiply<<<blocks, threads>>>(X, Y, Z, N);

    hipMemcpy(C, Z, matrixBytes, hipMemcpyDeviceToHost);
    printf("Multiplication of matrix A and B: \n");
    print(C, N);

    free(A);
    free(B);
    free(C);

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}
